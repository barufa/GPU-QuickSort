
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void multiplication(int * A,int * B,int * C,int N){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
	int col = blockIdx.x*blockDim.x+threadIdx.x;
	
	if(row<N && col<N){//Si no me fui del arreglo
		 int sum=0;
		 for(int i=0;i<N;i++){
			 sum+=A[row*N+i]*B[i*N+col];
		 }
		 C[row*N+col]=sum;
	}
}

#define N 30000

int main() {
    //Se creean los arreglos en el CPU
	int * A = (int *)malloc(sizeof(int)*(N*N));
	int * B = (int *)malloc(sizeof(int)*(N*N));
	int * C = (int *)malloc(sizeof(int)*(N*N));
	
	//~ srand(333);
	//~ for(int i=0;i<N;i++)for(int j=0;j<N;j++)A[i*N + j]=rand()%100;
	//~ for(int i=0;i<N;i++)for(int j=0;j<N;j++)B[i*N + j]=rand()%100;
	//~ for(int i=0;i<N;i++)for(int j=0;j<N;j++)C[i*N + j]=rand()%100;


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	
	//Se crean los arreglos en la GPU
	int *CU_A,*CU_B,*CU_C;
    hipMalloc((void **)&CU_A,sizeof(int)*(N*N));
    hipMalloc((void **)&CU_B,sizeof(int)*(N*N));
    hipMalloc((void **)&CU_C,sizeof(int)*(N*N));
	hipMemcpy(CU_A,A, N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(CU_B,B, N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(CU_C,C, N*N*sizeof(int), hipMemcpyHostToDevice);
	
	//Se ejecuta la funcion y se mide su performance
	multiplication<<<(N+255)/512,256>>>(CU_A,CU_B,CU_C,N);
	hipEventRecord(stop);
	
	//Se traen los resultados
	hipMemcpy(C,CU_C,N*sizeof(int),hipMemcpyDeviceToHost);
	
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	printf("Tiempo para matrices de %dx%d : %f ms.\n",N,N,milliseconds);
	
	//Validacion de los resultados
	//~ int t=1;
	//~ for(int i=0;i<N;i++)for(int j=0;j<N;j++){
		//~ int sum = 0;
		//~ for(int k=0;k<N;k++)sum+=A[i*N+k]*B[k*N+j];
		//~ if(C[i*N+j]!=sum)t=0;
	//~ }
	//~ if(t)puts("Algo salio mal");
	//~ else puts("YEAH!");
	
	//Se libera la memoria
	free(A);
	free(B);
	free(C);
	hipFree(CU_A);
	hipFree(CU_B);
	hipFree(CU_C);
	hipEventDestroy(start);
	hipEventDestroy(stop);
    return 0;
}

