
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#define SIZE 10000

__global__ void reduce(int * vector,int size,int pot){
	
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int salto = pot/2;
	
	while(salto){
		if(idx<salto && idx+salto<size){
			vector[idx]=vector[idx]+vector[idx+salto];
		} 
		__syncthreads();
		salto=salto/2;
	}
	
	return;
}

int main(void){
	
	int *N,*CU_N,len=SIZE,size=1;
	hipEvent_t start, stop;
	
	hipHostMalloc(&N,len*sizeof(int), hipHostMallocDefault);
	for(int i=0;i<len;i++)N[i]=1;
	while(size<len)size=(size<<1);
	
	puts("Arreglo inicializado");
	printf("Len:%d Size:%d\n",len,size);
	
	hipMalloc(&CU_N,len*sizeof(int));
	hipMemcpy(CU_N,N,len*sizeof(int),hipMemcpyHostToDevice);
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	reduce<<<(len+127)/128,128>>>(CU_N,len,size);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipMemcpy(N,CU_N,sizeof(int),hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Tiempo : %f ms.\n",milliseconds);
	printf("Resultado: %d\n",N[0]);
	//~ for(int i=0;i<len;i++)printf("%d%c",N[i]," \n"[i==len-1]);
	
	hipHostFree(N);
	hipFree(CU_N);
	
}


